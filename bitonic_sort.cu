#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 1024

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void bitonicSortShared(int* arr) {
    __shared__ int s_data[BLOCK_SIZE];
    int tid = threadIdx.x;
    int gid = blockIdx.x * BLOCK_SIZE + tid;

    s_data[tid] = arr[gid];
    __syncthreads();

    for (int k = 2; k <= BLOCK_SIZE; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            int ixj = tid ^ j;
            if (ixj > tid) {
                bool ascending = (tid & k) == 0;
                int val_i = s_data[tid];
                int val_j = s_data[ixj];

                if ((ascending && val_i > val_j) || (!ascending && val_i < val_j)) {
                    s_data[tid] = val_j;
                    s_data[ixj] = val_i;
                }
            }
            __syncthreads();
        }
    }

    arr[gid] = s_data[tid];
}

__global__ void bitonicMergeKernel(int* arr, int j, int k, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    int ixj = i ^ j;
    if (ixj > i && ixj < n) {
        bool ascending = (i & k) == 0;
        int val_i = arr[i];
        int val_j = arr[ixj];

        if ((ascending && val_i > val_j) || (!ascending && val_i < val_j)) {
            arr[i] = val_j;
            arr[ixj] = val_i;
        }
    }
}

// Utility function to run the full sort
void bitonicSort(int* h_arr, int n) {

    // cudaDeviceProp prop;
    // cudaGetDeviceProperties(&prop, 0);
    // printf("GPU: %s, SM count: %d, Shared memory per block: %zu\n", prop.name, prop.multiProcessorCount, prop.sharedMemPerBlock);

    // int minGridSize, blockSize;
    // cudaOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, bitonicSortShared, 0, 0);
    // printf("Suggested block size: %d\n", blockSize);

    int* d_arr;
    size_t bytes = n * sizeof(int);
    
    CUDA_CHECK(hipMalloc(&d_arr, bytes));
    CUDA_CHECK(hipMemcpy(d_arr, h_arr, bytes, hipMemcpyHostToDevice));
    
    // cudaMalloc(&d_arr, bytes);
    // cudaMemcpy(d_arr, h_arr, bytes, cudaMemcpyHostToDevice);
    // Step 1: Shared memory block-wise sort
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // printf("numBlocks=%d BLOCK_SIZE=%d\n", numBlocks, BLOCK_SIZE);
    bitonicSortShared<<<numBlocks, BLOCK_SIZE>>>(d_arr);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    // bitonicSortShared<<<numBlocks, BLOCK_SIZE>>>(d_arr);
    // cudaDeviceSynchronize();

    // Step 2: Global merge steps
    int threads = BLOCK_SIZE;
    int blocks = (n + threads - 1) / threads;

    for (int k = 2; k <= n; k <<= 1) { // k = 2, 4, 8, ...
        for (int j = k >> 1; j > 0; j >>= 1) { // j = k/2, k/4, ..., 1
            bitonicMergeKernel<<<blocks, threads>>>(d_arr, j, k, n);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(h_arr, d_arr, bytes, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}